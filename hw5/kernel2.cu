#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

/**
 * Calculates the number of iterations required for the given complex number to escape the Mandelbrot set.
 *
 * @param c_re The real part of the complex number.
 * @param c_im The imaginary part of the complex number.
 * @param count The maximum number of iterations to perform.
 * @return The number of iterations required for the complex number to escape the Mandelbrot set.
 */
__device__ static inline int mandel(float c_re, float c_im, int count) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(int *dev_output, float stepX, float stepY, float lowerX, float lowerY, int maxIterations, int pad) {
    // To avoid error caused by the floating number, use the following pseudo
    // code
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;
    int index = thisY * gridDim.x * blockDim.x + thisX;
    index += (index / (gridDim.x * blockDim.x)) * pad;
    dev_output[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Allocate memory for the output image
    // Allocate pinned host memory for the output image
    int* output;
    hipHostAlloc((void**)&output, resX * resY * sizeof(int), hipHostMallocDefault);

    // Allocate memory on the Cuda device
    int* dev_output;
    size_t pitch;
    hipMallocPitch((void**)&dev_output, &pitch, resX * sizeof(int), resY);

    // define the number of threads per block and the number of blocks
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((resX + threads_per_block.x - 1) / threads_per_block.x, (resY + threads_per_block.y - 1) / threads_per_block.y);

    // launch the kernel
    mandelKernel<<<num_blocks, threads_per_block>>>(dev_output, stepX, stepY, lowerX, lowerY, maxIterations, (pitch/sizeof(int)) - resX);

    // copy the output back to host
    hipMemcpy2D(output, resX * sizeof(int), dev_output, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

    // copy the output to the image
    memcpy(img, output, resX * resY * sizeof(int));
}
