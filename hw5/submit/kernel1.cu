#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/**
 * Allocates device memory using cudaMalloc and checks for errors.
 *
 * @param devPtr Pointer to the device memory to be allocated.
 * @param size   Size of the memory to be allocated.
 */
void safe_cuda_malloc(void** devPtr, size_t size) {
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Calculates the number of iterations required for the given complex number to escape the Mandelbrot set.
 *
 * @param c_re The real part of the complex number.
 * @param c_im The imaginary part of the complex number.
 * @param count The maximum number of iterations to perform.
 * @return The number of iterations required for the complex number to escape the Mandelbrot set.
 */
__device__ static inline int mandel(float c_re, float c_im, int count) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(int *dev_output, float stepX, float stepY, float lowerX, float lowerY, int maxIterations, int resX, int resY) {
    // To avoid error caused by the floating number, use the following pseudo
    // code
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;
    int index = thisY * gridDim.x * blockDim.x + thisX;
    
    if (thisX < 0 || thisX >= resX || thisY < 0 || thisY >= resY) return;
    dev_output[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Allocate memory for the output image
    int* output = (int*)malloc(resX * resY * sizeof(int));
    // Allocate memory on the Cuda device
    int* dev_output;
    safe_cuda_malloc((void**)&dev_output, resX * resY * sizeof(int));

    // define the number of threads per block and the number of blocks
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((resX + threads_per_block.x - 1) / threads_per_block.x, (resY + threads_per_block.y - 1) / threads_per_block.y);

    // launch the kernel
    mandelKernel<<<num_blocks, threads_per_block>>>(dev_output, stepX, stepY, lowerX, lowerY, maxIterations, resX, resY);

    // copy the output back to host
    hipMemcpy(output, dev_output, resX * resY * sizeof(int), hipMemcpyDeviceToHost);

    // copy the output to the image
    memcpy(img, output, resX * resY * sizeof(int));
}
