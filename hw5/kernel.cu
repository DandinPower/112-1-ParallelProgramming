#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

const int group_x_size = 16;
const int group_y_size = 12;

/**
 * Allocates device memory using cudaMalloc and checks for errors.
 *
 * @param devPtr Pointer to the device memory to be allocated.
 * @param size   Size of the memory to be allocated.
 */
void safe_cuda_malloc(void** devPtr, size_t size) {
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Calculates the number of iterations required for the given complex number to escape the Mandelbrot set.
 *
 * @param c_re The real part of the complex number.
 * @param c_im The imaginary part of the complex number.
 * @param count The maximum number of iterations to perform.
 * @return The number of iterations required for the complex number to escape the Mandelbrot set.
 */
__device__ static inline int mandel(float c_re, float c_im, int count) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(int *dev_output, float stepX, float stepY, float lowerX, float lowerY, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo
    // code
    // int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    // int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    int thread_x = (blockIdx.x * blockDim.x + threadIdx.x) * group_x_size;
    int thread_y = (blockIdx.y * blockDim.y + threadIdx.y) * group_y_size;

    int thisX, thisY;
    for (int i = 0; i < group_x_size; i++) {
        for (int j = 0; j < group_y_size; j++) {
            thisX = thread_x + i;
            thisY = thread_y + j;
            float x = lowerX + thisX * stepX;
            float y = lowerY + thisY * stepY;
            int answer = mandel(x, y, maxIterations);

            int index = thisY * gridDim.x * blockDim.x * group_x_size + thisX;
            
            dev_output[index] = answer;
        }

    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Allocate memory for the output image
    int* output = (int*)malloc(resX * resY * sizeof(int));
    // Allocate memory on the Cuda device
    int* dev_output;
    safe_cuda_malloc((void**)&dev_output, resX * resY * sizeof(int));

    // define the number of threads per block and the number of blocks
    dim3 threads_per_block(16, 16);
    dim3 num_blocks(((resX/group_x_size) + threads_per_block.x - 1) / threads_per_block.x, ((resY/group_y_size) + threads_per_block.y - 1) / threads_per_block.y);

    // launch the kernel
    mandelKernel<<<num_blocks, threads_per_block>>>(dev_output, stepX, stepY, lowerX, lowerY, maxIterations);

    // copy the output back to host
    hipMemcpy(output, dev_output, resX * resY * sizeof(int), hipMemcpyDeviceToHost);

    // copy the output to the image
    memcpy(img, output, resX * resY * sizeof(int));
}
