#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

const int group_x_size = 1;
const int group_y_size = 1;

/**
 * Allocates device memory using cudaMalloc and checks for errors.
 *
 * @param devPtr Pointer to the device memory to be allocated.
 * @param size   Size of the memory to be allocated.
 */
void safe_cuda_malloc(void** devPtr, size_t size) {
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Calculates the number of iterations required for the given complex number to escape the Mandelbrot set.
 *
 * @param c_re The real part of the complex number.
 * @param c_im The imaginary part of the complex number.
 * @param count The maximum number of iterations to perform.
 * @return The number of iterations required for the complex number to escape the Mandelbrot set.
 */
__device__ static inline int mandel(float c_re, float c_im, int count) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(int *dev_output, float stepX, float stepY, float lowerX, float lowerY, int maxIterations, int pad, int resX, int resY) {
    // To avoid error caused by the floating number, use the following pseudo
    // code
    // int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    // int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    int thread_x = (blockIdx.x * blockDim.x + threadIdx.x) * group_x_size;
    int thread_y = (blockIdx.y * blockDim.y + threadIdx.y) * group_y_size;

    for (int i = 0; i < group_x_size; i++) {
        for (int j = 0; j < group_y_size; j++) {
            int thisX = thread_x + i;
            int thisY = thread_y + j;

            float x = lowerX + thisX * stepX;
            float y = lowerY + thisY * stepY;
            int answer = mandel(x, y, maxIterations);

            int index = thisY * gridDim.x * blockDim.x * group_x_size + thisX;

            if (thisX >= resX || thisY >= resY) {
                continue;
            }
            if (index >= resX * resY) {
                continue;
            }

            index += (index / resX) * pad;
            dev_output[index] = answer;
        }

    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Allocate memory for the output image
    int* output;
    hipHostAlloc((void**)&output, resX * resY * sizeof(int), hipHostMallocDefault);
    // Allocate memory on the Cuda device
    int* dev_output;
    size_t pitch;
    hipMallocPitch((void**)&dev_output, &pitch, resX * sizeof(int), resY);

    // define the number of threads per block and the number of blocks
    dim3 threads_per_block(20, 20);
    dim3 num_blocks(((resX/group_x_size) + threads_per_block.x - 1) / threads_per_block.x, ((resY/group_y_size) + threads_per_block.y - 1) / threads_per_block.y);

    // launch the kernel
    mandelKernel<<<num_blocks, threads_per_block>>>(dev_output, stepX, stepY, lowerX, lowerY, maxIterations, (pitch/sizeof(int)) - resX, resX, resY);

    // wait for the kernel to finish
    // cudaDeviceSynchronize();

    // copy the output back to host
    hipMemcpy2D(output, resX * sizeof(int), dev_output, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

    // printf(
    //     "Total number of iterations: %d\n", output[0]
    // );

    // copy the output to the image
    memcpy(img, output, resX * resY * sizeof(int));
}
